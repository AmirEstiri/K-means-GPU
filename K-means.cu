#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
using namespace std;

#define BLOCK_SIZE 1024
#define n 100
#define K 10
#define d 3

struct Point{
float val[d];
};

__device__ float distance(Point* p1, Point* p2){
    float dist = 0;
    float tmp;
    for (int i = 0 ; i < d; i ++){
            tmp = (p1->val[i] - p2->val[i]);
            dist += tmp * tmp;
    }
    return dist;
}
__global__ void assignCluster(Point *data, int *cluster, Point *centers)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > n)
            return;

    int best_cluster = 0;
    float best_distance = 1e10;
    float tmp;

    for (int i = 0 ; i < K; i++){
            tmp = distance(data + i, centers + i);
            if (tmp > best_distance) {
                    best_distance = tmp;
                    best_cluster = i;
            }
    }
cluster[tid] = best_cluster;

}


__global__ void updateCenters(Point* data, int* cluster, Point* centers, int* cluster_sizes)
{

    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid >= n) return;

    int id = threadIdx.x;

    __shared__ Point data_shared[BLOCK_SIZE];
    __shared__ int cluster_shared[BLOCK_SIZE];
    __shared__ int cluster_sizes_shared[K];
    __shared__ Point cluster_sum[BLOCK_SIZE];

    if (id < K)
        cluster_sizes_shared[id] = 0;

    cluster_shared[id] = cluster[tid];
    data_shared[id]= data[tid];

    if (tid < K){
      for (int i = 0 ; i < d; i++)
          centers[tid].val[i] = 0;
        cluster_sizes[tid] = 0;
    }
__syncthreads();

for (int c = 0 ; c < K ; c++){
    if (cluster_shared[id] == c){
        for (int i = 0 ; i < d; i++)
            cluster_sum[id].val[i] = data_shared[id].val[i];
        atomicAdd(cluster_sizes_shared + c, 1);
    } else {
        for (int i = 0 ; i < d; i++)
            cluster_sum[id].val[i] = 0;
    }

    __syncthreads();

    for (int s = blockDim.x/2; s > 0; s /= 2) {
        if (id < s)
            for (int ind = 0; ind < d; ind++)
        cluster_sum[id].val[ind] += cluster_sum[id+s].val[ind];
        __syncthreads();
    }

    if (id == 0){
        for (int ind = 0; ind < d; ind++)
            centers[c].val[ind] += cluster_sum[0].val[ind];
        atomicAdd(cluster_sizes + c, cluster_sizes_shared[c]);
    }
}
__syncthreads();
    if(tid < K){
        for (int ind = 0; ind < d; ind++)
        centers[tid].val[ind] = centers[tid].val[ind]/cluster_sizes[tid];
    }

}


int main(void)
{
    Point *data;
    int *cluster;
    Point *centers;

    Point *d_data;
    int *d_cluster;
    Point *d_centers;

    int size_data = n * sizeof(Point);
    int size_cluster = n * sizeof(int);
    int size_centers = K * sizeof(Point);

    data = (Point*)malloc(size_data);
    cluster = (int*)malloc(size_cluster);
    centers = (Point*)malloc(size_centers);

    hipMalloc((void**)&d_data, size_data);
    hipMalloc((void**)&d_cluster, size_cluster);
hipMalloc((void**)&d_centers, size_centers);

int i, j;
for(i=0 ; i<n ; i++){
    for (j=0; j<d; j++)
        (data[i].val)[j] = i;
}

for(i=0 ; i<K ; i++){
    for (j=0; j<d; j++)
        (centers[i].val)[j] = (rand() % 10);
}

for (i=0; i<n; i++)
    cluster[i] = 0;

hipMemcpy(d_data, data, size_data, hipMemcpyHostToDevice);
hipMemcpy(d_centers, centers, size_centers, hipMemcpyHostToDevice);

dim3 dimBlock(n);
dim3 dimGrid(1);

// printf("%f\n", distance(data, data + 1));

assignCluster <<< dimGrid, dimBlock >>> (d_data, d_cluster, d_centers);
hipMemcpy( cluster, d_cluster, size_cluster, hipMemcpyDeviceToHost );
    return 0;
}