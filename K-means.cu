#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
using namespace std;
#define n 100
#define K 10
#define d 3

struct Point{
float val[d];
};

float distance(Point p1, Point p2){
float dist = 0;
float tmp;
for (int i = 0 ; i < d; i ++){
tmp = (p1->val[i] - p2->val[i]);
dist += tmp * tmp;
}
return dist;
}
_global_ void assignCluster(Point data, int cluster, Point *centers)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid > n)
return;

int best_cluster = 0;
float best_distance = 1e10;
float tmp;

for (int i = 0 ; i < K; i++){
tmp = distance(data[i], centers[i]);
if (tmp > best_distance) {
best_distance = tmp;
best_cluster = i;
}
}

cluster[tid] = best_cluster;

}


global void updateCenters(Point data, int clusters, Point centers, int cluster_sizes)
{

int tid = blockIdx.x*blockDim.x + threadIdx.x;
if (tid >= n) return;

int id = threadIdx.x;

shared Point data_shared[BLOCK_SIZE];
shared int clusters_shared[BLOCK_SIZE];
shared int clusters_sizes_shared[K];
shared Point cluster_sum[BLOCK_SIZE];

if (id < K)
clusters_sizes_shared[id] = 0;

clusters_shared[id] = clusters[tid];
data_shared[id]= data[tid];

if (tid < K){
centers[tid] = 0;
cluster_sizes[tid] = 0;
}

__syncthreads();

for (int c = 0 ; c < K ; c++){
if (clusters_shared[id] == c){
for (int i = 0 ; i < d; i++)
cluster_sum[id].var[i] = data_shared[id].var[i];
atomicAdd(clusters_sizes_shared + c, 1);
} else {
for (int i = 0 ; i < d; i++)
cluster_sum[id].var[i] = 0;
}

__syncthreads();

for (int s = blockDim.x/2; s > 0; s /= 2) {
if (id < s)
for (int ind = 0; ind < d; ind++)
cluster_sum[id].var[ind] += cluster_sum[id+s].var[ind];
__syncthreads();
}

if (id == 0){
for (int ind = 0; ind < d; ind++)
centers[c].var[ind] += cluster_sum[0].var[ind];
atomicAdd(cluster_sizes + c, cluster_sizes_shared[c]);
}
}
__syncthreads();

if(tid < K){
for (int ind = 0; ind < d; ind++)
centers[tid].var[ind] = centers[tid].var[ind]/cluster_sizes[tid];
}

}


int main(void)
{
Point *data;
int *cluster;
Point *centers;

Point *d_data;
int *d_cluster;
Point *d_centers;

int size_data = n * sizeof(Point);
int size_cluster = n * sizeof(int);
int size_centers = K * sizeof(Point);

data = (Point*)malloc(size_data);
cluster = (int*)malloc(size_cluster);
centers = (Point*)malloc(size_centers);

hipMalloc((void**)&d_data, size_data);
hipMalloc((void**)&d_cluster, size_cluster);
hipMalloc((void**)&d_centers, size_centers);

int i, j;
for(i=0 ; i<n ; i++){
for (j=0; j<d; j++)
(data[i]->val)[j] = i;
}

for(i=0 ; i<K ; i++){
for (j=0; j<d; j++)
(centers[i]->val)[j] = (rand() % 10);
}

for (i=0; i<n; i++)
cluster[i] = 0;

hipMemcpy(d_data, data, size_data, hipMemcpyHostToDevice);
hipMemcpy(d_centers, centers, size_centers, hipMemcpyHostToDevice);

//dim3 dimBlock(n);
//dim3 dimGrid(1);

printf("%f\n", distance(data, data + 1));

assignCluster <<< dimGrid, dimBlock >>> (d_data, d_cluster, d_centers);
hipMemcpy( cluster, d_cluster, size_cluster, hipMemcpyDeviceToHost );
return 0;
}